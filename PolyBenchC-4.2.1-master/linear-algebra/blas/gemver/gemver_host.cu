#include <assert.h>
#include <stdio.h>
#include "gemver_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gemver.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gemver.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE *alpha,
		 DATA_TYPE *beta,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		 DATA_TYPE POLYBENCH_1D(u1,N,n),
		 DATA_TYPE POLYBENCH_1D(v1,N,n),
		 DATA_TYPE POLYBENCH_1D(u2,N,n),
		 DATA_TYPE POLYBENCH_1D(v2,N,n),
		 DATA_TYPE POLYBENCH_1D(w,N,n),
		 DATA_TYPE POLYBENCH_1D(x,N,n),
		 DATA_TYPE POLYBENCH_1D(y,N,n),
		 DATA_TYPE POLYBENCH_1D(z,N,n))
{
  int i, j;

  *alpha = 1.5;
  *beta = 1.2;

  DATA_TYPE fn = (DATA_TYPE)n;

  for (i = 0; i < n; i++)
    {
      u1[i] = i;
      u2[i] = ((i+1)/fn)/2.0;
      v1[i] = ((i+1)/fn)/4.0;
      v2[i] = ((i+1)/fn)/6.0;
      y[i] = ((i+1)/fn)/8.0;
      z[i] = ((i+1)/fn)/9.0;
      x[i] = 0.0;
      w[i] = 0.0;
      for (j = 0; j < n; j++)
        A[i][j] = (DATA_TYPE) (i*j % n) / n;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(w,N,n))
{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("w");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, w[i]);
  }
  POLYBENCH_DUMP_END("w");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_gemver(int n,
		   DATA_TYPE alpha,
		   DATA_TYPE beta,
		   DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		   DATA_TYPE POLYBENCH_1D(u1,N,n),
		   DATA_TYPE POLYBENCH_1D(v1,N,n),
		   DATA_TYPE POLYBENCH_1D(u2,N,n),
		   DATA_TYPE POLYBENCH_1D(v2,N,n),
		   DATA_TYPE POLYBENCH_1D(w,N,n),
		   DATA_TYPE POLYBENCH_1D(x,N,n),
		   DATA_TYPE POLYBENCH_1D(y,N,n),
		   DATA_TYPE POLYBENCH_1D(z,N,n))
{
  int i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_u1;
    double *dev_u2;
    double *dev_v1;
    double *dev_v2;
    double *dev_w;
    double *dev_x;
    double *dev_y;
    double *dev_z;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (2000) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_u1, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_u2, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_v1, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_v2, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_w, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_x, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_y, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_z, (n) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (2000) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_u1, u1, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_u2, u2, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_v1, v1, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_v2, v2, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_w, w, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x, x, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y, y, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_z, z, (n) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(ppcg_min(256, (n + 31) / 32), ppcg_min(256, (n + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_u1, dev_u2, dev_v1, dev_v2, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(ppcg_min(32768, (n + 31) / 32));
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, beta, dev_x, dev_y, dev_z, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k2_dimBlock(32);
      dim3 k2_dimGrid(ppcg_min(32768, (n + 31) / 32));
      kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_A, alpha, dev_w, dev_x, n);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(A, dev_A, (n) * (2000) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(w, dev_w, (n) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x, dev_x, (n) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_u1));
    cudaCheckReturn(hipFree(dev_u2));
    cudaCheckReturn(hipFree(dev_v1));
    cudaCheckReturn(hipFree(dev_v2));
    cudaCheckReturn(hipFree(dev_w));
    cudaCheckReturn(hipFree(dev_x));
    cudaCheckReturn(hipFree(dev_y));
    cudaCheckReturn(hipFree(dev_z));
  }
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(u1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(v1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(u2, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(v2, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(w, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(z, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n, &alpha, &beta,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(u1),
	      POLYBENCH_ARRAY(v1),
	      POLYBENCH_ARRAY(u2),
	      POLYBENCH_ARRAY(v2),
	      POLYBENCH_ARRAY(w),
	      POLYBENCH_ARRAY(x),
	      POLYBENCH_ARRAY(y),
	      POLYBENCH_ARRAY(z));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gemver (n, alpha, beta,
		 POLYBENCH_ARRAY(A),
		 POLYBENCH_ARRAY(u1),
		 POLYBENCH_ARRAY(v1),
		 POLYBENCH_ARRAY(u2),
		 POLYBENCH_ARRAY(v2),
		 POLYBENCH_ARRAY(w),
		 POLYBENCH_ARRAY(x),
		 POLYBENCH_ARRAY(y),
		 POLYBENCH_ARRAY(z));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(w)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(u1);
  POLYBENCH_FREE_ARRAY(v1);
  POLYBENCH_FREE_ARRAY(u2);
  POLYBENCH_FREE_ARRAY(v2);
  POLYBENCH_FREE_ARRAY(w);
  POLYBENCH_FREE_ARRAY(x);
  POLYBENCH_FREE_ARRAY(y);
  POLYBENCH_FREE_ARRAY(z);

  return 0;
}
