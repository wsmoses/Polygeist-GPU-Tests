#include "hip/hip_runtime.h"
#include "gemver_kernel.hu"
__global__ void kernel0(double *A, double *u1, double *u2, double *v1, double *v2, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_u1[32];
    __shared__ double shared_u2[32];
    __shared__ double shared_v1[32];
    __shared__ double shared_v2[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 8192) {
      if (t0 == 0) {
        for (int c1 = t1; c1 <= ppcg_min(31, n - c0 - 1); c1 += 16)
          shared_u1[c1] = u1[c0 + c1];
        for (int c1 = t1; c1 <= ppcg_min(31, n - c0 - 1); c1 += 16)
          shared_u2[c1] = u2[c0 + c1];
      }
      __syncthreads();
      for (int c1 = 32 * b1; c1 < n; c1 += 8192) {
        if (t0 == 0) {
          for (int c2 = t1; c2 <= ppcg_min(31, n - c1 - 1); c2 += 16)
            shared_v1[c2] = v1[c1 + c2];
          for (int c2 = t1; c2 <= ppcg_min(31, n - c1 - 1); c2 += 16)
            shared_v2[c2] = v2[c1 + c2];
        }
        __syncthreads();
        if (n >= t0 + c0 + 1)
          for (int c3 = t1; c3 <= ppcg_min(31, n - c1 - 1); c3 += 16)
            A[(t0 + c0) * 2000 + (c1 + c3)] = ((A[(t0 + c0) * 2000 + (c1 + c3)] + (shared_u1[t0] * shared_v1[c3])) + (shared_u2[t0] * shared_v2[c3]));
        __syncthreads();
      }
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double beta, double *x, double *y, double *z, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_x[1];
    __shared__ double shared_y[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      if (n >= t0 + c0 + 1)
        private_x[0] = x[t0 + c0];
      for (int c1 = 0; c1 <= n; c1 += 32) {
        if (n >= t0 + c1 + 1)
          shared_y[t0] = y[t0 + c1];
        __syncthreads();
        if (n >= t0 + c0 + 1) {
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_x[0] = (private_x[0] + ((beta * A[(c1 + c3) * 2000 + (t0 + c0)]) * shared_y[c3]));
          if (c1 + 31 >= n)
            private_x[0] = (private_x[0] + z[t0 + c0]);
        }
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        x[t0 + c0] = private_x[0];
      __syncthreads();
    }
}
__global__ void kernel2(double *A, double alpha, double *w, double *x, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_w[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      if (n >= t0 + c0 + 1)
        private_w[0] = w[t0 + c0];
      for (int c1 = 0; c1 < n; c1 += 32) {
        if (t0 + c1 <= 1999)
          for (int c2 = 0; c2 <= ppcg_min(31, n - c0 - 1); c2 += 1)
            shared_A[c2][t0] = A[(c0 + c2) * 2000 + (t0 + c1)];
        __syncthreads();
        if (n >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_w[0] = (private_w[0] + ((alpha * shared_A[t0][c3]) * x[c1 + c3]));
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        w[t0 + c0] = private_w[0];
      __syncthreads();
    }
}
