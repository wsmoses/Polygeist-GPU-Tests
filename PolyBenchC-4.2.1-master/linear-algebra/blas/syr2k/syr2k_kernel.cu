#include "hip/hip_runtime.h"
#include "syr2k_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int n, int m)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A_0[32][32];
    double private_C[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * ((b0 - b1 + 256) % 256) + 32 * b1; c0 < n; c0 += 8192)
      for (int c1 = 32 * b1; c1 <= ppcg_min(n - 1, c0 + 31); c1 += 8192) {
        if (b1 <= 37 && n >= t0 + c0 + 1 && t0 + c0 >= 32 * b1 + t1 && c1 == 32 * b1) {
          private_C[0][0] = C[(t0 + c0) * 1200 + (32 * b1 + t1)];
          if (32 * b1 + t1 <= 1183 && t0 + c0 >= 32 * b1 + t1 + 16)
            private_C[0][1] = C[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)];
        }
        for (int c2 = 0; c2 < m; c2 += 32) {
          if (n >= t0 + c1 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 999); c4 += 16)
              shared_A_0[t0][c4] = A[(t0 + c1) * 1000 + (c2 + c4)];
          __syncthreads();
          if (n >= t0 + c0 + 1 && t0 + c0 >= t1 + c1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (t0 + c0 >= t1 + c1 + 16)
              private_C[0][1] *= beta;
          }
          if (n >= t0 + c0 + 1 && t0 + c0 >= t1 + c1)
            for (int c3 = 0; c3 <= ppcg_min(31, m - c2 - 1); c3 += 1) {
              private_C[0][0] += (((shared_A_0[t1][c3] * alpha) * B[(t0 + c0) * 1000 + (c2 + c3)]) + ((B[(t1 + c1) * 1000 + (c2 + c3)] * alpha) * A[(t0 + c0) * 1000 + (c2 + c3)]));
              if (t0 + c0 >= t1 + c1 + 16)
                private_C[0][1] += (((shared_A_0[t1 + 16][c3] * alpha) * B[(t0 + c0) * 1000 + (c2 + c3)]) + ((B[(t1 + c1 + 16) * 1000 + (c2 + c3)] * alpha) * A[(t0 + c0) * 1000 + (c2 + c3)]));
            }
          __syncthreads();
        }
        if (m <= 0) {
          __syncthreads();
          if (n >= t0 + c0 + 1 && t0 + c0 >= t1 + c1) {
            private_C[0][0] *= beta;
            if (t0 + c0 >= t1 + c1 + 16)
              private_C[0][1] *= beta;
          }
          __syncthreads();
        }
        if (b1 <= 37 && n >= t0 + c0 + 1 && t0 + c0 >= 32 * b1 + t1 && c1 == 32 * b1) {
          C[(t0 + c0) * 1200 + (32 * b1 + t1)] = private_C[0][0];
          if (32 * b1 + t1 <= 1183 && t0 + c0 >= 32 * b1 + t1 + 16)
            C[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)] = private_C[0][1];
        }
        __syncthreads();
      }
}
