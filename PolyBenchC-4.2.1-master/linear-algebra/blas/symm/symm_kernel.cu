#include "hip/hip_runtime.h"
#include "symm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A_1[32][32];
    double private_temp2;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 8192) {
      if (m >= 32 * b0 + t0 + 1 && c0 == 32 * b0)
        for (int c2 = t1; c2 <= ppcg_min(31, -32 * b0 + 999); c2 += 16)
          shared_A_1[t0][c2] = A[(32 * b0 + t0) * 1000 + (32 * b0 + c2)];
      __syncthreads();
      if (m >= t0 + c0 + 1)
        for (int c1 = 32 * b1; c1 < n; c1 += 8192)
          for (int c3 = t1; c3 <= ppcg_min(31, n - c1 - 1); c3 += 16) {
            private_temp2 = 0;
            for (int c4 = 0; c4 < t0 + c0; c4 += 1)
              private_temp2 += (B[c4 * 1200 + (c1 + c3)] * A[(t0 + c0) * 1000 + c4]);
            C[(t0 + c0) * 1200 + (c1 + c3)] = (((beta * C[(t0 + c0) * 1200 + (c1 + c3)]) + ((alpha * B[(t0 + c0) * 1200 + (c1 + c3)]) * shared_A_1[t0][t0])) + (alpha * private_temp2));
          }
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double *B, double *C, double alpha, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_C[2][1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 8192)
      for (int c1 = 32 * b1; c1 < m - 1; c1 += 8192) {
        if (n >= 32 * b0 + t0 + 1 && 32 * b0 + t0 <= 1199 && c0 == 32 * b0 && m >= t1 + c1 + 2) {
          private_C[0][0] = C[(t1 + c1) * 1200 + (32 * b0 + t0)];
          if (m >= t1 + c1 + 18)
            private_C[1][0] = C[(t1 + c1 + 16) * 1200 + (32 * b0 + t0)];
        }
        for (int c2 = c1; c2 < m - 1; c2 += 32) {
          if (c1 == 32 * b1 && m >= t0 + c2 + 2)
            for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 999); c4 += 16)
              shared_A[t0][c4] = A[(t0 + c2 + 1) * 1000 + (32 * b1 + c4)];
          __syncthreads();
          if (n >= t0 + c0 + 1)
            for (int c3 = ppcg_max(0, t1 + c1 - c2); c3 <= ppcg_min(31, m - c2 - 2); c3 += 1) {
              private_C[0][0] += ((alpha * B[(c2 + c3 + 1) * 1200 + (t0 + c0)]) * shared_A[c3][t1]);
              if (c2 + c3 >= t1 + c1 + 16)
                private_C[1][0] += ((alpha * B[(c2 + c3 + 1) * 1200 + (t0 + c0)]) * shared_A[c3][t1 + 16]);
            }
          __syncthreads();
        }
        if (n >= 32 * b0 + t0 + 1 && 32 * b0 + t0 <= 1199 && c0 == 32 * b0 && m >= t1 + c1 + 2) {
          C[(t1 + c1) * 1200 + (32 * b0 + t0)] = private_C[0][0];
          if (m >= t1 + c1 + 18)
            C[(t1 + c1 + 16) * 1200 + (32 * b0 + t0)] = private_C[1][0];
        }
        __syncthreads();
      }
}
