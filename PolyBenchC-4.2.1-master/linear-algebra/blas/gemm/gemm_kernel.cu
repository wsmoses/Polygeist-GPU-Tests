#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int ni, int nk, int nj)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_C[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (nj >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 1099 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          private_C[0][0] = C[(t0 + c0) * 1100 + (32 * b1 + t1)];
          if (nj >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1083)
            private_C[0][1] = C[(t0 + c0) * 1100 + (32 * b1 + t1 + 16)];
        }
        for (int c2 = 0; c2 < nk; c2 += 32) {
          if (ni >= t0 + c0 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1199); c4 += 16)
              shared_A[t0][c4] = A[(t0 + c0) * 1200 + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17)
              private_C[0][1] *= beta;
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * 1100 + (t1 + c1)]);
              if (nj >= t1 + c1 + 17)
                private_C[0][1] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * 1100 + (t1 + c1 + 16)]);
            }
          __syncthreads();
        }
        if (nk <= 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17)
              private_C[0][1] *= beta;
          }
          __syncthreads();
        }
        if (nj >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 1099 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          C[(t0 + c0) * 1100 + (32 * b1 + t1)] = private_C[0][0];
          if (nj >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1083)
            C[(t0 + c0) * 1100 + (32 * b1 + t1 + 16)] = private_C[0][1];
        }
        __syncthreads();
      }
}
