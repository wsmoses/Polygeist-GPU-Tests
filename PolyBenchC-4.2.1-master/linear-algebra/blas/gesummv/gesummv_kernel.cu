#include "hip/hip_runtime.h"
#include "gesummv_kernel.hu"
__global__ void kernel0(double *A, double *B, double alpha, double beta, double *tmp, double *x, double *y, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_tmp[1];
    double private_y[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      for (int c1 = 0; c1 <= n; c1 += 32) {
        if (t0 + c1 <= 1299)
          for (int c2 = 0; c2 <= ppcg_min(31, n - c0 - 1); c2 += 1)
            shared_A[c2][t0] = A[(c0 + c2) * 1300 + (t0 + c1)];
        __syncthreads();
        if (n >= t0 + c0 + 1 && c1 == 0) {
          private_y[0] = 0.;
          private_tmp[0] = 0.;
        }
        if (n >= t0 + c0 + 1) {
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1) {
            private_y[0] = ((B[(t0 + c0) * 1300 + (c1 + c3)] * x[c1 + c3]) + private_y[0]);
            private_tmp[0] = ((shared_A[t0][c3] * x[c1 + c3]) + private_tmp[0]);
          }
          if (c1 + 31 >= n)
            private_y[0] = ((alpha * private_tmp[0]) + (beta * private_y[0]));
        }
        __syncthreads();
      }
      if (n >= t0 + c0 + 1) {
        y[t0 + c0] = private_y[0];
        tmp[t0 + c0] = private_tmp[0];
      }
      __syncthreads();
    }
}
