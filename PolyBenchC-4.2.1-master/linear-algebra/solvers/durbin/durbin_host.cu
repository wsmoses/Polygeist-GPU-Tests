#include <assert.h>
#include <stdio.h>
#include "durbin_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* durbin.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "durbin.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_1D(r,N,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      r[i] = (n+1-i);
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(y,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("y");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, y[i]);
  }
  POLYBENCH_DUMP_END("y");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_durbin(int n,
		   DATA_TYPE POLYBENCH_1D(r,N,n),
		   DATA_TYPE POLYBENCH_1D(y,N,n))
{
 DATA_TYPE z[N];
 DATA_TYPE alpha;
 DATA_TYPE beta;
 DATA_TYPE sum;

 int i,k;

