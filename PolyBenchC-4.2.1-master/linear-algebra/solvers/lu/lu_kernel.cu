#include "hip/hip_runtime.h"
#include "lu_kernel.hu"
__global__ void kernel0(double *A, int n, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A_1[1][1];

    {
      if (t0 == 0 && c0 <= 1999)
        shared_A_1[0][0] = A[c0 * 2000 + c0];
      __syncthreads();
      for (int c1 = 32 * b0 + 1048576 * ((-32 * b0 + c0 + 1048544) / 1048576); c1 < n - 1; c1 += 1048576)
        if (n >= t0 + c1 + 2 && t0 + c1 >= c0)
          A[(t0 + c1 + 1) * 2000 + c0] /= shared_A_1[0][0];
    }
}
__global__ void kernel1(double *A, int n, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A_1[32][1];
    __shared__ double shared_A_2[1][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c1 = 32 * b0 + 8192 * ((-32 * b0 + c0 + 8160) / 8192); c1 < 32 * ((b0 - b1 + 255) % 256) + n - 8161; c1 += 8192) {
      if (t1 == 0 && c0 <= 1999 && n >= t0 + c1 + 2)
        shared_A_1[t0][0] = A[(t0 + c1 + 1) * 2000 + c0];
      __syncthreads();
      for (int c2 = 32 * b1 + 8192 * ((-32 * b1 + c1 + 8160) / 8192); c2 < n - 1; c2 += 8192) {
        if (t0 == 0 && c1 == 32 * b0 && c2 == 32 * b1)
          for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1998); c4 += 16)
            shared_A_2[0][c4] = A[c0 * 2000 + (32 * b1 + c4 + 1)];
        __syncthreads();
        if (t0 + c1 >= c0)
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c1 - c2 - 1, 16) + 16); c4 <= ppcg_min(31, n - c2 - 2); c4 += 16)
            A[(t0 + c1 + 1) * 2000 + (c2 + c4 + 1)] -= (shared_A_1[t0][0] * shared_A_2[0][c4]);
        __syncthreads();
      }
      __syncthreads();
    }
}
__global__ void kernel2(double *A, int n, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A_1[32][1];
    __shared__ double shared_A_2[1][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c1 = 32 * ((b0 - b1 + 256) % 256) - ((-32 * b1 + c0 + 8160) % 8192) + c0 + 8160; c1 < n - 2; c1 += 8192) {
      if (t1 == 0 && c0 <= 1999 && n >= t0 + c1 + 3)
        shared_A_1[t0][0] = A[(t0 + c1 + 2) * 2000 + c0];
      __syncthreads();
      for (int c2 = 32 * b1 + 8192 * ((-32 * b1 + c0 + 8160) / 8192); c2 <= ppcg_min(n - 3, c1 + 31); c2 += 8192) {
        if (t0 == 0 && c2 == 32 * b1)
          for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1998); c4 += 16)
            shared_A_2[0][c4] = A[c0 * 2000 + (32 * b1 + c4 + 1)];
        __syncthreads();
        if (n >= t0 + c1 + 3)
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(-t1 + c0 - c2 - 1, 16) + 16); c4 <= ppcg_min(31, t0 + c1 - c2); c4 += 16)
            A[(t0 + c1 + 2) * 2000 + (c2 + c4 + 1)] -= (shared_A_1[t0][0] * shared_A_2[0][c4]);
        __syncthreads();
      }
      __syncthreads();
    }
}
