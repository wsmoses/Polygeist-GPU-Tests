#include "hip/hip_runtime.h"
#include "3mm_kernel.hu"
__global__ void kernel0(double *C, double *D, double *F, int ni, int nl, int nj, int nm, int nk)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_C[32][32];
    double private_F[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < nj; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nl; c1 += 8192) {
        for (int c2 = 0; c2 < nm; c2 += 32) {
          if (nj >= t0 + c0 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1199); c4 += 16)
              shared_C[t0][c4] = C[(t0 + c0) * 1200 + (c2 + c4)];
          __syncthreads();
          if (nj >= t0 + c0 + 1 && nl >= t1 + c1 + 1 && c2 == 0) {
            private_F[0][0] = 0.;
            if (nl >= t1 + c1 + 17)
              private_F[0][1] = 0.;
          }
          if (nj >= t0 + c0 + 1 && nl >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nm - c2 - 1); c3 += 1) {
              private_F[0][0] += (shared_C[t0][c3] * D[(c2 + c3) * 1100 + (t1 + c1)]);
              if (nl >= t1 + c1 + 17)
                private_F[0][1] += (shared_C[t0][c3] * D[(c2 + c3) * 1100 + (t1 + c1 + 16)]);
            }
          __syncthreads();
        }
        if (nm <= 0) {
          __syncthreads();
          if (nj >= t0 + c0 + 1 && nl >= t1 + c1 + 1) {
            private_F[0][0] = 0.;
            if (nl >= t1 + c1 + 17)
              private_F[0][1] = 0.;
          }
          __syncthreads();
        }
        if (nl >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 1099 && nj >= t0 + c0 + 1 && c1 == 32 * b1) {
          F[(t0 + c0) * 1100 + (32 * b1 + t1)] = private_F[0][0];
          if (nl >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1083)
            F[(t0 + c0) * 1100 + (32 * b1 + t1 + 16)] = private_F[0][1];
        }
        __syncthreads();
      }
}
__global__ void kernel1(double *A, double *B, double *E, int ni, int nl, int nj, int nm, int nk)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_E[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        for (int c2 = 0; c2 < nk; c2 += 32) {
          if (ni >= t0 + c0 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 999); c4 += 16)
              shared_A[t0][c4] = A[(t0 + c0) * 1000 + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_E[0][0] = 0.;
            if (nj >= t1 + c1 + 17)
              private_E[0][1] = 0.;
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_E[0][0] += (shared_A[t0][c3] * B[(c2 + c3) * 900 + (t1 + c1)]);
              if (nj >= t1 + c1 + 17)
                private_E[0][1] += (shared_A[t0][c3] * B[(c2 + c3) * 900 + (t1 + c1 + 16)]);
            }
          __syncthreads();
        }
        if (nk <= 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_E[0][0] = 0.;
            if (nj >= t1 + c1 + 17)
              private_E[0][1] = 0.;
          }
          __syncthreads();
        }
        if (nj >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 899 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          E[(t0 + c0) * 900 + (32 * b1 + t1)] = private_E[0][0];
          if (nj >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 883)
            E[(t0 + c0) * 900 + (32 * b1 + t1 + 16)] = private_E[0][1];
        }
        __syncthreads();
      }
}
__global__ void kernel2(double *E, double *F, double *G, int ni, int nl, int nj, int nm, int nk)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_E[32][32];
    double private_G[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nl; c1 += 8192) {
        if (nj >= 1 && ni >= t0 + c0 + 1)
          for (int c4 = t1; c4 <= 31; c4 += 16)
            shared_E[t0][c4] = E[(t0 + c0) * 900 + c4];
        __syncthreads();
        if (ni >= t0 + c0 + 1 && nl >= t1 + c1 + 1) {
          private_G[0][0] = 0.;
          if (nl >= t1 + c1 + 17)
            private_G[0][1] = 0.;
          for (int c3 = 0; c3 <= ppcg_min(31, nj - 1); c3 += 1) {
            private_G[0][0] += (shared_E[t0][c3] * F[c3 * 1100 + (t1 + c1)]);
            if (nl >= t1 + c1 + 17)
              private_G[0][1] += (shared_E[t0][c3] * F[c3 * 1100 + (t1 + c1 + 16)]);
          }
        }
        __syncthreads();
        for (int c2 = 32; c2 < nj; c2 += 32) {
          if (ni >= t0 + c0 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 899); c4 += 16)
              shared_E[t0][c4] = E[(t0 + c0) * 900 + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nl >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nj - c2 - 1); c3 += 1) {
              private_G[0][0] += (shared_E[t0][c3] * F[(c2 + c3) * 1100 + (t1 + c1)]);
              if (nl >= t1 + c1 + 17)
                private_G[0][1] += (shared_E[t0][c3] * F[(c2 + c3) * 1100 + (t1 + c1 + 16)]);
            }
          __syncthreads();
        }
        if (nl >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 1099 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          G[(t0 + c0) * 1100 + (32 * b1 + t1)] = private_G[0][0];
          if (nl >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1083)
            G[(t0 + c0) * 1100 + (32 * b1 + t1 + 16)] = private_G[0][1];
        }
        __syncthreads();
      }
}
