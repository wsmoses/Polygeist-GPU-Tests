#include "hip/hip_runtime.h"
#include "mvt_kernel.hu"
__global__ void kernel0(double *A, double *x2, double *y_2, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_x2[1];
    __shared__ double shared_y_2[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      if (n >= t0 + c0 + 1)
        private_x2[0] = x2[t0 + c0];
      for (int c1 = 0; c1 < n; c1 += 32) {
        if (n >= t0 + c1 + 1)
          shared_y_2[t0] = y_2[t0 + c1];
        __syncthreads();
        if (n >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_x2[0] = (private_x2[0] + (A[(c1 + c3) * 2000 + (t0 + c0)] * shared_y_2[c3]));
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        x2[t0 + c0] = private_x2[0];
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double *x1, double *y_1, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_x1[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      if (n >= t0 + c0 + 1)
        private_x1[0] = x1[t0 + c0];
      for (int c1 = 0; c1 < n; c1 += 32) {
        if (t0 + c1 <= 1999)
          for (int c2 = 0; c2 <= ppcg_min(31, n - c0 - 1); c2 += 1)
            shared_A[c2][t0] = A[(c0 + c2) * 2000 + (t0 + c1)];
        __syncthreads();
        if (n >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_x1[0] = (private_x1[0] + (shared_A[t0][c3] * y_1[c1 + c3]));
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        x1[t0 + c0] = private_x1[0];
      __syncthreads();
    }
}
