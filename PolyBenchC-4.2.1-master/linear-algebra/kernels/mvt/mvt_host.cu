#include <assert.h>
#include <stdio.h>
#include "mvt_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* mvt.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "mvt.h"


/* Array initialization. */
static
void init_array(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      x1[i] = (DATA_TYPE) (i % n) / n;
      x2[i] = (DATA_TYPE) ((i + 1) % n) / n;
      y_1[i] = (DATA_TYPE) ((i + 3) % n) / n;
      y_2[i] = (DATA_TYPE) ((i + 4) % n) / n;
      for (j = 0; j < n; j++)
	A[i][j] = (DATA_TYPE) (i*j % n) / n;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(x1,N,n),
		 DATA_TYPE POLYBENCH_1D(x2,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("x1");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x1[i]);
  }
  POLYBENCH_DUMP_END("x1");

  POLYBENCH_DUMP_BEGIN("x2");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x2[i]);
  }
  POLYBENCH_DUMP_END("x2");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_mvt(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_x1;
    double *dev_x2;
    double *dev_y_1;
    double *dev_y_2;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (2000) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_x1, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_x2, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_1, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_2, (n) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (2000) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x1, x1, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x2, x2, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_1, y_1, (n) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_2, y_2, (n) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(ppcg_min(32768, (n + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_x2, dev_y_2, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(ppcg_min(32768, (n + 31) / 32));
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_x1, dev_y_1, n);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(x1, dev_x1, (n) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x2, dev_x2, (n) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_x1));
    cudaCheckReturn(hipFree(dev_x2));
    cudaCheckReturn(hipFree(dev_y_1));
    cudaCheckReturn(hipFree(dev_y_2));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(x1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x2, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_2, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_mvt (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(x1);
  POLYBENCH_FREE_ARRAY(x2);
  POLYBENCH_FREE_ARRAY(y_1);
  POLYBENCH_FREE_ARRAY(y_2);

  return 0;
}
