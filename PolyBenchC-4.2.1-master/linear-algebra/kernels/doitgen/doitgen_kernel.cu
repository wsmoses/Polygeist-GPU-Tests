#include "hip/hip_runtime.h"
#include "doitgen_kernel.hu"
__global__ void kernel0(double *A, double *C4, double *sum, int nr, int nq, int np, int c0, int c1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[1][1][32];
    double private_sum[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 32 * b0; c2 < np; c2 += 1048576) {
      for (int c3 = 0; c3 < np; c3 += 32) {
        if (c1 <= 139 && t0 + c3 <= 159)
          shared_A[0][0][t0] = A[(c0 * 140 + c1) * 160 + (t0 + c3)];
        __syncthreads();
        if (np >= t0 + c2 + 1 && c3 == 0)
          private_sum[0] = 0.;
        if (np >= t0 + c2 + 1)
          for (int c5 = 0; c5 <= ppcg_min(31, np - c3 - 1); c5 += 1)
            private_sum[0] += (shared_A[0][0][c5] * C4[(c3 + c5) * 160 + (t0 + c2)]);
        __syncthreads();
      }
      if (np >= t0 + c2 + 1)
        sum[t0 + c2] = private_sum[0];
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double *sum, int nr, int nq, int np, int c0, int c1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c2 = 32 * b0; c2 < np; c2 += 1048576)
      if (np >= t0 + c2 + 1)
        A[(c0 * 140 + c1) * 160 + (t0 + c2)] = sum[t0 + c2];
}
