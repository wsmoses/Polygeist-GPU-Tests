#include "hip/hip_runtime.h"
#include "2mm_kernel.hu"
__global__ void kernel0(double *A, double *B, double alpha, double *tmp, int ni, int nl, int nj, int nk)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_tmp[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        for (int c2 = 0; c2 < nk; c2 += 32) {
          if (ni >= t0 + c0 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1099); c4 += 16)
              shared_A[t0][c4] = A[(t0 + c0) * 1100 + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_tmp[0][0] = 0.;
            if (nj >= t1 + c1 + 17)
              private_tmp[0][1] = 0.;
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_tmp[0][0] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * 900 + (t1 + c1)]);
              if (nj >= t1 + c1 + 17)
                private_tmp[0][1] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * 900 + (t1 + c1 + 16)]);
            }
          __syncthreads();
        }
        if (nk <= 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_tmp[0][0] = 0.;
            if (nj >= t1 + c1 + 17)
              private_tmp[0][1] = 0.;
          }
          __syncthreads();
        }
        if (nj >= 32 * b1 + t1 + 1 && 32 * b1 + t1 <= 899 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          tmp[(t0 + c0) * 900 + (32 * b1 + t1)] = private_tmp[0][0];
          if (nj >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 883)
            tmp[(t0 + c0) * 900 + (32 * b1 + t1 + 16)] = private_tmp[0][1];
        }
        __syncthreads();
      }
}
__global__ void kernel1(double *C, double *D, double beta, double *tmp, int ni, int nl, int nj, int nk)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_C[32][32];
    double private_D[1][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nl; c1 += 8192) {
        if (b1 <= 37 && nl >= 32 * b1 + t1 + 1 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          private_D[0][0] = D[(t0 + c0) * 1200 + (32 * b1 + t1)];
          if (nl >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
            private_D[0][1] = D[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)];
        }
        for (int c2 = 0; c2 < nj; c2 += 32) {
          if (c1 == 32 * b1 && nj >= t0 + c2 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1199); c4 += 16)
              shared_C[t0][c4] = C[(t0 + c2) * 1200 + (32 * b1 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nl >= t1 + c1 + 1 && c2 == 0) {
            private_D[0][0] *= beta;
            if (nl >= t1 + c1 + 17)
              private_D[0][1] *= beta;
          }
          if (ni >= t0 + c0 + 1 && nl >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nj - c2 - 1); c3 += 1) {
              private_D[0][0] += (tmp[(t0 + c0) * 900 + (c2 + c3)] * shared_C[c3][t1]);
              if (nl >= t1 + c1 + 17)
                private_D[0][1] += (tmp[(t0 + c0) * 900 + (c2 + c3)] * shared_C[c3][t1 + 16]);
            }
          __syncthreads();
        }
        if (nj <= 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nl >= t1 + c1 + 1) {
            private_D[0][0] *= beta;
            if (nl >= t1 + c1 + 17)
              private_D[0][1] *= beta;
          }
          __syncthreads();
        }
        if (b1 <= 37 && nl >= 32 * b1 + t1 + 1 && ni >= t0 + c0 + 1 && c1 == 32 * b1) {
          D[(t0 + c0) * 1200 + (32 * b1 + t1)] = private_D[0][0];
          if (nl >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
            D[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)] = private_D[0][1];
        }
        __syncthreads();
      }
}
