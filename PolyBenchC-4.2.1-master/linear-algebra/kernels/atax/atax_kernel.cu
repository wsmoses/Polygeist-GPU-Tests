#include "hip/hip_runtime.h"
#include "atax_kernel.hu"
__global__ void kernel0(double *A, double *tmp, double *x, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_tmp[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 1048576) {
      if (n >= 1)
        for (int c2 = 0; c2 <= ppcg_min(31, m - c0 - 1); c2 += 1)
          shared_A[c2][t0] = A[(c0 + c2) * 2100 + t0];
      __syncthreads();
      if (m >= t0 + c0 + 1) {
        private_tmp[0] = 0.;
        for (int c3 = 0; c3 <= ppcg_min(31, n - 1); c3 += 1)
          private_tmp[0] = (private_tmp[0] + (shared_A[t0][c3] * x[c3]));
      }
      __syncthreads();
      for (int c1 = 32; c1 < n; c1 += 32) {
        if (t0 + c1 <= 2099)
          for (int c2 = 0; c2 <= ppcg_min(31, m - c0 - 1); c2 += 1)
            shared_A[c2][t0] = A[(c0 + c2) * 2100 + (t0 + c1)];
        __syncthreads();
        if (m >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_tmp[0] = (private_tmp[0] + (shared_A[t0][c3] * x[c1 + c3]));
        __syncthreads();
      }
      if (m >= t0 + c0 + 1)
        tmp[t0 + c0] = private_tmp[0];
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double *tmp, double *y, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_tmp[32];
    double private_y[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      for (int c1 = 0; c1 < m; c1 += 32) {
        if (m >= t0 + c1 + 1)
          shared_tmp[t0] = tmp[t0 + c1];
        __syncthreads();
        if (n >= t0 + c0 + 1 && c1 == 0)
          private_y[0] = 0;
        if (n >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, m - c1 - 1); c3 += 1)
            private_y[0] = (private_y[0] + (A[(c1 + c3) * 2100 + (t0 + c0)] * shared_tmp[c3]));
        __syncthreads();
      }
      if (m <= 0) {
        __syncthreads();
        if (n >= t0 + c0 + 1)
          private_y[0] = 0;
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        y[t0 + c0] = private_y[0];
      __syncthreads();
    }
}
