#include <assert.h>
#include <stdio.h>
#include "bicg_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* bicg.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "bicg.h"


/* Array initialization. */
static
void init_array (int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m))
{
  int i, j;

  for (i = 0; i < m; i++)
    p[i] = (DATA_TYPE)(i % m) / m;
  for (i = 0; i < n; i++) {
    r[i] = (DATA_TYPE)(i % n) / n;
    for (j = 0; j < m; j++)
      A[i][j] = (DATA_TYPE) (i*(j+1) % n)/n;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m, int n,
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("s");
  for (i = 0; i < m; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, s[i]);
  }
  POLYBENCH_DUMP_END("s");
  POLYBENCH_DUMP_BEGIN("q");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, q[i]);
  }
  POLYBENCH_DUMP_END("q");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_bicg(int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n))
{
  int i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (m >= 1 || n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_p;
    double *dev_q;
    double *dev_r;
    double *dev_s;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (1900) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_p, (m) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_q, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_r, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_s, (m) * sizeof(double)));
    
    if (n >= 1 && m >= 1) {
      cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (1900) * sizeof(double), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_p, p, (m) * sizeof(double), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_r, r, (n) * sizeof(double), hipMemcpyHostToDevice));
    }
    if (m >= 1)
      {
        dim3 k0_dimBlock(32);
        dim3 k0_dimGrid(ppcg_min(32768, (m + 31) / 32));
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_r, dev_s, n, m);
        cudaCheckKernel();
      }
      
    if (n >= 1) {
      {
        dim3 k1_dimBlock(32);
        dim3 k1_dimGrid(ppcg_min(32768, (n + 31) / 32));
        kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_p, dev_q, n, m);
        cudaCheckKernel();
      }
      
      cudaCheckReturn(hipMemcpy(q, dev_q, (n) * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (m >= 1)
      cudaCheckReturn(hipMemcpy(s, dev_s, (m) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_p));
    cudaCheckReturn(hipFree(dev_q));
    cudaCheckReturn(hipFree(dev_r));
    cudaCheckReturn(hipFree(dev_s));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, M, n, m);
  POLYBENCH_1D_ARRAY_DECL(s, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(q, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(p, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(r, DATA_TYPE, N, n);

  /* Initialize array(s). */
  init_array (m, n,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(r),
	      POLYBENCH_ARRAY(p));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_bicg (m, n,
	       POLYBENCH_ARRAY(A),
	       POLYBENCH_ARRAY(s),
	       POLYBENCH_ARRAY(q),
	       POLYBENCH_ARRAY(p),
	       POLYBENCH_ARRAY(r));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, n, POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(q)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(s);
  POLYBENCH_FREE_ARRAY(q);
  POLYBENCH_FREE_ARRAY(p);
  POLYBENCH_FREE_ARRAY(r);

  return 0;
}
