#include "hip/hip_runtime.h"
#include "bicg_kernel.hu"
__global__ void kernel0(double *A, double *r, double *s, int n, int m)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_r[32];
    double private_s[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 1048576) {
      if (n >= t0 + 1)
        shared_r[t0] = r[t0];
      __syncthreads();
      if (m >= t0 + c0 + 1) {
        private_s[0] = 0;
        for (int c3 = 0; c3 <= ppcg_min(31, n - 1); c3 += 1)
          private_s[0] = (private_s[0] + (shared_r[c3] * A[c3 * 1900 + (t0 + c0)]));
      }
      __syncthreads();
      for (int c1 = 32; c1 < n; c1 += 32) {
        if (n >= t0 + c1 + 1)
          shared_r[t0] = r[t0 + c1];
        __syncthreads();
        if (m >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_s[0] = (private_s[0] + (shared_r[c3] * A[(c1 + c3) * 1900 + (t0 + c0)]));
        __syncthreads();
      }
      if (m >= t0 + c0 + 1)
        s[t0 + c0] = private_s[0];
      __syncthreads();
    }
}
__global__ void kernel1(double *A, double *p, double *q, int n, int m)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_q[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 1048576) {
      for (int c1 = 0; c1 < m; c1 += 32) {
        if (t0 + c1 <= 1899)
          for (int c2 = 0; c2 <= ppcg_min(31, n - c0 - 1); c2 += 1)
            shared_A[c2][t0] = A[(c0 + c2) * 1900 + (t0 + c1)];
        __syncthreads();
        if (n >= t0 + c0 + 1 && c1 == 0)
          private_q[0] = 0.;
        if (n >= t0 + c0 + 1)
          for (int c3 = 0; c3 <= ppcg_min(31, m - c1 - 1); c3 += 1)
            private_q[0] = (private_q[0] + (shared_A[t0][c3] * p[c1 + c3]));
        __syncthreads();
      }
      if (m <= 0) {
        __syncthreads();
        if (n >= t0 + c0 + 1)
          private_q[0] = 0.;
        __syncthreads();
      }
      if (n >= t0 + c0 + 1)
        q[t0 + c0] = private_q[0];
      __syncthreads();
    }
}
