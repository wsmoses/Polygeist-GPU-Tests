#include "hip/hip_runtime.h"
#include "jacobi-1d_kernel.hu"
__global__ void kernel0(double *A, double *B, int tsteps, int n, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < n - 2; c1 += 1048576)
      if (n >= t0 + c1 + 3)
        B[t0 + c1 + 1] = (0.33333 * ((A[t0 + c1] + A[t0 + c1 + 1]) + A[t0 + c1 + 2]));
}
__global__ void kernel1(double *A, double *B, int tsteps, int n, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < n - 2; c1 += 1048576)
      if (n >= t0 + c1 + 3)
        A[t0 + c1 + 1] = (0.33333 * ((B[t0 + c1] + B[t0 + c1 + 1]) + B[t0 + c1 + 2]));
}
