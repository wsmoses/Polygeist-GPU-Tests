#include "hip/hip_runtime.h"
#include "jacobi-2d_kernel.hu"
__global__ void kernel0(double *A, double *B, int tsteps, int n, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < n - 2; c1 += 8192)
      if (n >= t0 + c1 + 3)
        for (int c2 = 32 * b1; c2 < n - 2; c2 += 8192)
          for (int c4 = t1; c4 <= ppcg_min(31, n - c2 - 3); c4 += 16)
            B[(t0 + c1 + 1) * 1300 + (c2 + c4 + 1)] = (0.20000000000000001 * ((((A[(t0 + c1 + 1) * 1300 + (c2 + c4 + 1)] + A[(t0 + c1 + 1) * 1300 + (c2 + c4)]) + A[(t0 + c1 + 1) * 1300 + (c2 + c4 + 2)]) + A[(t0 + c1 + 2) * 1300 + (c2 + c4 + 1)]) + A[(t0 + c1) * 1300 + (c2 + c4 + 1)]));
}
__global__ void kernel1(double *A, double *B, int tsteps, int n, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < n - 2; c1 += 8192)
      if (n >= t0 + c1 + 3)
        for (int c2 = 32 * b1; c2 < n - 2; c2 += 8192)
          for (int c4 = t1; c4 <= ppcg_min(31, n - c2 - 3); c4 += 16)
            A[(t0 + c1 + 1) * 1300 + (c2 + c4 + 1)] = (0.20000000000000001 * ((((B[(t0 + c1 + 1) * 1300 + (c2 + c4 + 1)] + B[(t0 + c1 + 1) * 1300 + (c2 + c4)]) + B[(t0 + c1 + 1) * 1300 + (c2 + c4 + 2)]) + B[(t0 + c1 + 2) * 1300 + (c2 + c4 + 1)]) + B[(t0 + c1) * 1300 + (c2 + c4 + 1)]));
}
