#include "hip/hip_runtime.h"
#include "seidel-2d_kernel.hu"
__global__ void kernel0(double *A, int tsteps, int n, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c1 = ppcg_max(32 * b0, 32 * b0 + 8192 * ppcg_fdiv_q(-3 * n - 128 * b0 + c0 - 119, 32768) + 8192); c1 < ppcg_min(tsteps, -16 * b1 + (c0 + 1) / 4); c1 += 8192)
      if (tsteps >= t0 + c1 + 1)
        for (int c2 = ppcg_max(ppcg_max(32 * b1, 32 * b1 + 8192 * ppcg_fdiv_q(-4 * tsteps - n - 64 * b1 + c0 - 59, 16384) + 8192), 32 * b1 + 8192 * ppcg_fdiv_q(-n - 64 * b1 + c0 - 4 * c1 - 187, 16384) + 8192); c2 < ppcg_min(n - 2, -2 * c1 + (c0 + 1) / 2 - 1); c2 += 8192)
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(-n - 4 * t0 - 2 * t1 + c0 - 4 * c1 - 2 * c2 - 1, 32) + 16); c4 <= ppcg_min(ppcg_min(31, n - c2 - 3), -2 * t0 - 2 * c1 - c2 + (c0 + 1) / 2 - 2); c4 += 16)
            A[(c2 + c4 + 1) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 2)] = (((((((((A[(c2 + c4) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 3)] + A[(c2 + c4) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 2)]) + A[(c2 + c4) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 1)]) + A[(c2 + c4 + 1) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 3)]) + A[(c2 + c4 + 1) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 2)]) + A[(c2 + c4 + 1) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 1)]) + A[(c2 + c4 + 2) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 3)]) + A[(c2 + c4 + 2) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 2)]) + A[(c2 + c4 + 2) * 2000 + (-4 * t0 + c0 - 4 * c1 - 2 * c2 - 2 * c4 - 1)]) / 9.);
}
