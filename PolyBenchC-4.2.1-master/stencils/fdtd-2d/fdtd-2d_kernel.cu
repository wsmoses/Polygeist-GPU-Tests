#include "hip/hip_runtime.h"
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double *ey, double *hz, int tmax, int nx, int ny, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < nx - 1; c1 += 8192)
      if (nx >= t0 + c1 + 2)
        for (int c2 = 32 * b1; c2 < ny; c2 += 8192)
          for (int c4 = t1; c4 <= ppcg_min(31, ny - c2 - 1); c4 += 16)
            ey[(t0 + c1 + 1) * 1200 + (c2 + c4)] = (ey[(t0 + c1 + 1) * 1200 + (c2 + c4)] - (0.5 * (hz[(t0 + c1 + 1) * 1200 + (c2 + c4)] - hz[(t0 + c1) * 1200 + (c2 + c4)])));
}
__global__ void kernel1(double *ex, double *hz, int tmax, int nx, int ny, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < nx; c1 += 8192)
      if (nx >= t0 + c1 + 1)
        for (int c2 = 32 * b1; c2 < ny - 1; c2 += 8192)
          for (int c4 = t1; c4 <= ppcg_min(31, ny - c2 - 2); c4 += 16)
            ex[(t0 + c1) * 1200 + (c2 + c4 + 1)] = (ex[(t0 + c1) * 1200 + (c2 + c4 + 1)] - (0.5 * (hz[(t0 + c1) * 1200 + (c2 + c4 + 1)] - hz[(t0 + c1) * 1200 + (c2 + c4)])));
}
__global__ void kernel2(double *_fict_, double *ex, double *ey, double *hz, int tmax, int nx, int ny, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared__fict_[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (t0 == 0 && t1 == 0)
        shared__fict_[0] = _fict_[c0];
      __syncthreads();
      if (ny >= 32 * b1 + 2)
        for (int c1 = 32 * b0; c1 < nx - 1; c1 += 8192)
          if (nx >= t0 + c1 + 2) {
            for (int c2 = 32 * b1; c2 < ny - 1; c2 += 8192)
              if (ny >= t1 + c2 + 1) {
                for (int c4 = t1; c4 <= ppcg_min(31, ny - c2 - 2); c4 += 16) {
                  if (b0 == 0 && t0 == 0 && c1 == 0)
                    ey[0 * 1200 + (c2 + c4)] = shared__fict_[0];
                  hz[(t0 + c1) * 1200 + (c2 + c4)] = (hz[(t0 + c1) * 1200 + (c2 + c4)] - (0.69999999999999996 * (((ex[(t0 + c1) * 1200 + (c2 + c4 + 1)] - ex[(t0 + c1) * 1200 + (c2 + c4)]) + ey[(t0 + c1 + 1) * 1200 + (c2 + c4)]) - ey[(t0 + c1) * 1200 + (c2 + c4)])));
                }
                if (b0 == 0 && t0 == 0 && c1 == 0 && c2 + 32 >= ny && (ny - t1 - 1) % 16 == 0)
                  ey[0 * 1200 + (ny - 1)] = shared__fict_[0];
              }
            if (b0 == 0 && t0 == 0 && t1 == 0 && c1 == 0 && (-ny + 32 * b1 + 1) % 8192 == 0)
              ey[0 * 1200 + (ny - 1)] = shared__fict_[0];
          }
      if (nx <= 1 && b0 == 0 && t0 == 0) {
        for (int c2 = 32 * b1; c2 < ny; c2 += 8192)
          for (int c4 = t1; c4 <= ppcg_min(31, ny - c2 - 1); c4 += 16)
            ey[0 * 1200 + (c2 + c4)] = shared__fict_[0];
      } else if (nx >= 2 && b0 == 0 && 32 * b1 + 1 == ny && t0 == 0 && t1 == 0) {
        ey[0 * 1200 + (ny - 1)] = shared__fict_[0];
      }
    }
}
