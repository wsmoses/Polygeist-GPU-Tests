#include <assert.h>
#include <stdio.h>
#include "fdtd-2d_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* fdtd-2d.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "fdtd-2d.h"


/* Array initialization. */
static
void init_array (int tmax,
		 int nx,
		 int ny,
		 DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_1D(_fict_,TMAX,tmax))
{
  int i, j;

  for (i = 0; i < tmax; i++)
    _fict_[i] = (DATA_TYPE) i;
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
      {
	ex[i][j] = ((DATA_TYPE) i*(j+1)) / nx;
	ey[i][j] = ((DATA_TYPE) i*(j+2)) / ny;
	hz[i][j] = ((DATA_TYPE) i*(j+3)) / nx;
      }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx,
		 int ny,
		 DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		 DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("ex");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, ex[i][j]);
    }
  POLYBENCH_DUMP_END("ex");
  POLYBENCH_DUMP_FINISH;

  POLYBENCH_DUMP_BEGIN("ey");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, ey[i][j]);
    }
  POLYBENCH_DUMP_END("ey");

  POLYBENCH_DUMP_BEGIN("hz");
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      if ((i * nx + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, hz[i][j]);
    }
  POLYBENCH_DUMP_END("hz");
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_fdtd_2d(int tmax,
		    int nx,
		    int ny,
		    DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny),
		    DATA_TYPE POLYBENCH_1D(_fict_,TMAX,tmax))
{
  int t, i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (tmax >= 1 && ny >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev__fict_;
    double *dev_ex;
    double *dev_ey;
    double *dev_hz;
    
    cudaCheckReturn(hipMalloc((void **) &dev__fict_, (tmax) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_ex, (nx) * (1200) * sizeof(double)));
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    cudaCheckReturn(hipMalloc((void **) &dev_ey, (ppcg_max(1, nx)) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_hz, (nx) * (1200) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev__fict_, _fict_, (tmax) * sizeof(double), hipMemcpyHostToDevice));
    if (nx >= 1 && ny >= 2)
      cudaCheckReturn(hipMemcpy(dev_ex, ex, (nx) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_ey, ey, (ppcg_max(1, nx)) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    if (nx >= 1 && nx + ny >= 3)
      cudaCheckReturn(hipMemcpy(dev_hz, hz, (nx) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    for (int c0 = 0; c0 < tmax; c0 += 1) {
      if (nx >= 2)
        {
          dim3 k0_dimBlock(16, 32);
          dim3 k0_dimGrid(ppcg_min(256, (ny + 31) / 32), ppcg_min(256, (nx + 30) / 32));
          kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_ey, dev_hz, tmax, nx, ny, c0);
          cudaCheckKernel();
        }
        
      if (nx >= 1 && ny >= 2)
        {
          dim3 k1_dimBlock(16, 32);
          dim3 k1_dimGrid(ppcg_min(256, (ny + 30) / 32), ppcg_min(256, (nx + 31) / 32));
          kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_ex, dev_hz, tmax, nx, ny, c0);
          cudaCheckKernel();
        }
        
      {
        dim3 k2_dimBlock(16, 32);
        dim3 k2_dimGrid(ppcg_min(256, (ny + 31) / 32), nx >= 34 && ny >= 2 ? ppcg_min(256, (nx + 30) / 32) : 1);
        kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev__fict_, dev_ex, dev_ey, dev_hz, tmax, nx, ny, c0);
        cudaCheckKernel();
      }
      
    }
    if (nx >= 1 && ny >= 2)
      cudaCheckReturn(hipMemcpy(ex, dev_ex, (nx) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(ey, dev_ey, (ppcg_max(1, nx)) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    if (nx >= 1 && nx + ny >= 3)
      cudaCheckReturn(hipMemcpy(hz, dev_hz, (nx) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev__fict_));
    cudaCheckReturn(hipFree(dev_ex));
    cudaCheckReturn(hipFree(dev_ey));
    cudaCheckReturn(hipFree(dev_hz));
  }
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int tmax = TMAX;
  int nx = NX;
  int ny = NY;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(ex,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_2D_ARRAY_DECL(ey,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_2D_ARRAY_DECL(hz,DATA_TYPE,NX,NY,nx,ny);
  POLYBENCH_1D_ARRAY_DECL(_fict_,DATA_TYPE,TMAX,tmax);

  /* Initialize array(s). */
  init_array (tmax, nx, ny,
	      POLYBENCH_ARRAY(ex),
	      POLYBENCH_ARRAY(ey),
	      POLYBENCH_ARRAY(hz),
	      POLYBENCH_ARRAY(_fict_));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_fdtd_2d (tmax, nx, ny,
		  POLYBENCH_ARRAY(ex),
		  POLYBENCH_ARRAY(ey),
		  POLYBENCH_ARRAY(hz),
		  POLYBENCH_ARRAY(_fict_));


  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(nx, ny, POLYBENCH_ARRAY(ex),
				    POLYBENCH_ARRAY(ey),
				    POLYBENCH_ARRAY(hz)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(ex);
  POLYBENCH_FREE_ARRAY(ey);
  POLYBENCH_FREE_ARRAY(hz);
  POLYBENCH_FREE_ARRAY(_fict_);

  return 0;
}
