#include "hip/hip_runtime.h"
#include "covariance_kernel.hu"
__global__ void kernel0(double *data, double *mean, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_mean[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 1048576) {
      if (m >= t0 + c0 + 1) {
        private_mean[0] = 0.;
        for (int c1 = 0; c1 < n; c1 += 32)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_mean[0] += data[(c1 + c3) * 1200 + (t0 + c0)];
        mean[t0 + c0] = private_mean[0];
      }
      __syncthreads();
    }
}
__global__ void kernel1(double *cov, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c0 = 32 * b0; c0 < m; c0 += 8192)
      for (int c1 = 32 * b1 + 8192 * ((-32 * b1 + c0 + 8160) / 8192); c1 < m; c1 += 8192)
        for (int c3 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c0 - c1 - 1, 16) + 16); c3 <= ppcg_min(31, m - c1 - 1); c3 += 16)
          cov[(t0 + c0) * 1200 + (c1 + c3)] = 0.;
}
__global__ void kernel2(double float_n, double *mean, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c0 = 32 * b0; c0 < m; c0 += 1048576)
      if (m >= t0 + c0 + 1)
        mean[t0 + c0] /= float_n;
}
__global__ void kernel3(double *data, double *mean, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_mean[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 8192)
      for (int c1 = 32 * b1; c1 < m; c1 += 8192) {
        if (t0 == 0)
          for (int c2 = t1; c2 <= ppcg_min(31, m - c1 - 1); c2 += 16)
            shared_mean[c2] = mean[c1 + c2];
        __syncthreads();
        if (n >= t0 + c0 + 1)
          for (int c3 = t1; c3 <= ppcg_min(31, m - c1 - 1); c3 += 16)
            data[(t0 + c0) * 1200 + (c1 + c3)] -= shared_mean[c3];
        __syncthreads();
      }
}
__global__ void kernel4(double *cov, double *data, double float_n, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_data_0[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c0 = 32 * b0; c0 < m; c0 += 8192)
      for (int c1 = 32 * b1 + 8192 * ((-32 * b1 + c0 + 8160) / 8192); c1 < m; c1 += 8192) {
        for (int c2 = 0; c2 < n; c2 += 32) {
          if (c0 == 32 * b0 && n >= t0 + c2 + 1)
            for (int c4 = t1; c4 <= ppcg_min(31, -32 * b0 + 1199); c4 += 16)
              shared_data_0[t0][c4] = data[(t0 + c2) * 1200 + (32 * b0 + c4)];
          __syncthreads();
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c0 - c1 - 1, 16) + 16); c4 <= ppcg_min(31, m - c1 - 1); c4 += 16) {
            for (int c5 = 0; c5 <= ppcg_min(31, n - c2 - 1); c5 += 1)
              cov[(t0 + c0) * 1200 + (c1 + c4)] += (shared_data_0[c5][t0] * data[(c2 + c5) * 1200 + (c1 + c4)]);
            if (c2 + 31 >= n) {
              cov[(t0 + c0) * 1200 + (c1 + c4)] /= (float_n - 1.);
              cov[(c1 + c4) * 1200 + (t0 + c0)] = cov[(t0 + c0) * 1200 + (c1 + c4)];
            }
          }
          __syncthreads();
        }
        if (n >= 32 && n % 32 == 0) {
          __syncthreads();
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c0 - c1 - 1, 16) + 16); c4 <= ppcg_min(31, m - c1 - 1); c4 += 16) {
            cov[(t0 + c0) * 1200 + (c1 + c4)] /= (float_n - 1.);
            cov[(c1 + c4) * 1200 + (t0 + c0)] = cov[(t0 + c0) * 1200 + (c1 + c4)];
          }
          __syncthreads();
        }
        if (n <= 0) {
          __syncthreads();
          for (int c4 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c0 - c1 - 1, 16) + 16); c4 <= ppcg_min(31, m - c1 - 1); c4 += 16) {
            cov[(t0 + c0) * 1200 + (c1 + c4)] /= (float_n - 1.);
            cov[(c1 + c4) * 1200 + (t0 + c0)] = cov[(t0 + c0) * 1200 + (c1 + c4)];
          }
          __syncthreads();
        }
      }
}
