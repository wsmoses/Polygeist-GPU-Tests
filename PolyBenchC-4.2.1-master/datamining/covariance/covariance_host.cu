#include <assert.h>
#include <stdio.h>
#include "covariance_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* covariance.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "covariance.h"


/* Array initialization. */
static
void init_array (int m, int n,
		 DATA_TYPE *float_n,
		 DATA_TYPE POLYBENCH_2D(data,N,M,n,m))
{
  int i, j;

  *float_n = (DATA_TYPE)n;

  for (i = 0; i < N; i++)
    for (j = 0; j < M; j++)
      data[i][j] = ((DATA_TYPE) i*j) / M;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m,
		 DATA_TYPE POLYBENCH_2D(cov,M,M,m,m))

{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("cov");
  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
      if ((i * m + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
      fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, cov[i][j]);
    }
  POLYBENCH_DUMP_END("cov");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_covariance(int m, int n,
		       DATA_TYPE float_n,
		       DATA_TYPE POLYBENCH_2D(data,N,M,n,m),
		       DATA_TYPE POLYBENCH_2D(cov,M,M,m,m),
		       DATA_TYPE POLYBENCH_1D(mean,M,m))
{
  int i, j, k;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (m >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_cov;
    double *dev_data;
    double *dev_mean;
    
    cudaCheckReturn(hipMalloc((void **) &dev_cov, (m) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_data, (n) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_mean, (m) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev_cov, cov, (m) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    if (n >= 1)
      cudaCheckReturn(hipMemcpy(dev_data, data, (n) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(ppcg_min(32768, (m + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_data, dev_mean, m, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(16, 32);
      dim3 k1_dimGrid(ppcg_min(256, (m + 31) / 32), ppcg_min(256, (m + 31) / 32));
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_cov, m, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k2_dimBlock(32);
      dim3 k2_dimGrid(ppcg_min(32768, (m + 31) / 32));
      kernel2 <<<k2_dimGrid, k2_dimBlock>>> (float_n, dev_mean, m, n);
      cudaCheckKernel();
    }
    
    if (n >= 1)
      {
        dim3 k3_dimBlock(16, 32);
        dim3 k3_dimGrid(ppcg_min(256, (m + 31) / 32), ppcg_min(256, (n + 31) / 32));
        kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_data, dev_mean, m, n);
        cudaCheckKernel();
      }
      
    {
      dim3 k4_dimBlock(16, 32);
      dim3 k4_dimGrid(ppcg_min(256, (m + 31) / 32), ppcg_min(256, (m + 31) / 32));
      kernel4 <<<k4_dimGrid, k4_dimBlock>>> (dev_cov, dev_data, float_n, m, n);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(cov, dev_cov, (m) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    if (n >= 1)
      cudaCheckReturn(hipMemcpy(data, dev_data, (n) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(mean, dev_mean, (m) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_cov));
    cudaCheckReturn(hipFree(dev_data));
    cudaCheckReturn(hipFree(dev_mean));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  DATA_TYPE float_n;
  POLYBENCH_2D_ARRAY_DECL(data,DATA_TYPE,N,M,n,m);
  POLYBENCH_2D_ARRAY_DECL(cov,DATA_TYPE,M,M,m,m);
  POLYBENCH_1D_ARRAY_DECL(mean,DATA_TYPE,M,m);


  /* Initialize array(s). */
  init_array (m, n, &float_n, POLYBENCH_ARRAY(data));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_covariance (m, n, float_n,
		     POLYBENCH_ARRAY(data),
		     POLYBENCH_ARRAY(cov),
		     POLYBENCH_ARRAY(mean));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, POLYBENCH_ARRAY(cov)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(data);
  POLYBENCH_FREE_ARRAY(cov);
  POLYBENCH_FREE_ARRAY(mean);

  return 0;
}
