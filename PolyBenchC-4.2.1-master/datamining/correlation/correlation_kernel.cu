#include "hip/hip_runtime.h"
#include "correlation_kernel.hu"
__global__ void kernel0(double *corr, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c0 = 32 * b0; c0 < m - 1; c0 += 1048576)
      if (m >= t0 + c0 + 2)
        corr[(t0 + c0) * 1200 + (t0 + c0)] = 1.;
}
__global__ void kernel1(double *data, double *mean, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_mean[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 1048576) {
      if (m >= t0 + c0 + 1) {
        private_mean[0] = 0.;
        for (int c1 = 0; c1 < n; c1 += 32)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_mean[0] += data[(c1 + c3) * 1200 + (t0 + c0)];
        mean[t0 + c0] = private_mean[0];
      }
      __syncthreads();
    }
}
__global__ void kernel2(double *data, double float_n, double *mean, double *stddev, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_mean[1];
    double private_stddev[1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m; c0 += 1048576) {
      if (m >= t0 + c0 + 1) {
        private_mean[0] = mean[t0 + c0];
        private_mean[0] /= float_n;
        private_stddev[0] = 0.;
        for (int c1 = 0; c1 < n; c1 += 32)
          for (int c3 = 0; c3 <= ppcg_min(31, n - c1 - 1); c3 += 1)
            private_stddev[0] += ((data[(c1 + c3) * 1200 + (t0 + c0)] - private_mean[0]) * (data[(c1 + c3) * 1200 + (t0 + c0)] - private_mean[0]));
        stddev[t0 + c0] = private_stddev[0];
        mean[t0 + c0] = private_mean[0];
      }
      __syncthreads();
    }
}
__global__ void kernel3(double eps, double float_n, double *stddev, int m, int n)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    double private_stddev[1];

    for (int c0 = 32 * b0; c0 < m; c0 += 1048576)
      if (m >= t0 + c0 + 1) {
        private_stddev[0] = stddev[t0 + c0];
        private_stddev[0] /= float_n;
        private_stddev[0] = sqrt(private_stddev[0]);
        private_stddev[0] = ((private_stddev[0] <= eps) ? 1. : private_stddev[0]);
        stddev[t0 + c0] = private_stddev[0];
      }
}
__global__ void kernel4(double *data, double float_n, double *mean, double *stddev, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_data[1][2];
    __shared__ double shared_mean[32];
    __shared__ double shared_stddev[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < n; c0 += 8192)
      for (int c1 = 32 * b1; c1 < m; c1 += 8192) {
        if (b1 <= 37 && m >= 32 * b1 + t1 + 1 && n >= t0 + c0 + 1 && c1 == 32 * b1) {
          private_data[0][0] = data[(t0 + c0) * 1200 + (32 * b1 + t1)];
          if (m >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
            private_data[0][1] = data[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)];
        }
        if (t0 == 0) {
          for (int c2 = t1; c2 <= ppcg_min(31, m - c1 - 1); c2 += 16)
            shared_mean[c2] = mean[c1 + c2];
          for (int c2 = t1; c2 <= ppcg_min(31, m - c1 - 1); c2 += 16)
            shared_stddev[c2] = stddev[c1 + c2];
        }
        __syncthreads();
        if (n >= t0 + c0 + 1 && m >= t1 + c1 + 1) {
          private_data[0][0] -= shared_mean[t1];
          if (m >= t1 + c1 + 17)
            private_data[0][1] -= shared_mean[t1 + 16];
          private_data[0][0] /= (sqrt(float_n) * shared_stddev[t1]);
          if (m >= t1 + c1 + 17)
            private_data[0][1] /= (sqrt(float_n) * shared_stddev[t1 + 16]);
        }
        __syncthreads();
        if (b1 <= 37 && m >= 32 * b1 + t1 + 1 && n >= t0 + c0 + 1 && c1 == 32 * b1) {
          data[(t0 + c0) * 1200 + (32 * b1 + t1)] = private_data[0][0];
          if (m >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
            data[(t0 + c0) * 1200 + (32 * b1 + t1 + 16)] = private_data[0][1];
        }
      }
}
__global__ void kernel5(double *corr, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c0 = 32 * b0; c0 < m - 1; c0 += 8192)
      for (int c1 = 32 * b1 + 8192 * ((-32 * b1 + c0 + 8160) / 8192); c1 < m - 1; c1 += 8192)
        for (int c3 = ppcg_max(t1, t1 + 16 * ppcg_fdiv_q(t0 - t1 + c0 - c1 - 1, 16) + 16); c3 <= ppcg_min(31, m - c1 - 2); c3 += 16)
          corr[(t0 + c0) * 1200 + (c1 + c3 + 1)] = 0.;
}
__global__ void kernel6(double *corr, double *data, int m, int n)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_corr_0[1][2];
    __shared__ double shared_corr_1[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < m - 1; c0 += 8192)
      for (int c1 = 32 * b1 + 8192 * ((-32 * b1 + c0 + 8161) / 8192); c1 < m; c1 += 8192) {
        if (b1 <= 37 && m >= 32 * b0 + t0 + 2 && 32 * b0 + t0 <= 1198 && m >= 32 * b1 + t1 + 1 && 32 * b1 + t1 + 15 >= 32 * b0 + t0 && c0 == 32 * b0 && c1 == 32 * b1) {
          if (32 * b1 + t1 >= 32 * b0 + t0 + 1)
            private_corr_0[0][0] = corr[(32 * b0 + t0) * 1200 + (32 * b1 + t1)];
          if (m >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
            private_corr_0[0][1] = corr[(32 * b0 + t0) * 1200 + (32 * b1 + t1 + 16)];
        }
        __syncthreads();
        if (m >= t0 + c0 + 2 && m >= t1 + c1 + 1 && t1 + c1 + 15 >= t0 + c0) {
          for (int c2 = 0; c2 < n; c2 += 32)
            for (int c3 = 0; c3 <= ppcg_min(31, n - c2 - 1); c3 += 1) {
              if (t1 + c1 >= t0 + c0 + 1)
                private_corr_0[0][0] += (data[(c2 + c3) * 1200 + (t0 + c0)] * data[(c2 + c3) * 1200 + (t1 + c1)]);
              if (m >= t1 + c1 + 17)
                private_corr_0[0][1] += (data[(c2 + c3) * 1200 + (t0 + c0)] * data[(c2 + c3) * 1200 + (t1 + c1 + 16)]);
            }
          if (t1 + c1 >= t0 + c0 + 1)
            shared_corr_1[t1][t0] = private_corr_0[0][0];
          if (m >= t1 + c1 + 17)
            shared_corr_1[t1 + 16][t0] = private_corr_0[0][1];
          if (n >= 1 && b1 <= 37 && 32 * b0 + t0 <= 1198 && c0 == 32 * b0 && c1 == 32 * b1) {
            if (32 * b1 + t1 >= 32 * b0 + t0 + 1)
              corr[(32 * b0 + t0) * 1200 + (32 * b1 + t1)] = private_corr_0[0][0];
            if (m >= 32 * b1 + t1 + 17 && 32 * b1 + t1 <= 1183)
              corr[(32 * b0 + t0) * 1200 + (32 * b1 + t1 + 16)] = private_corr_0[0][1];
          }
        }
        __syncthreads();
        if (m >= t0 + c1 + 1)
          for (int c3 = t1; c3 <= ppcg_min(31, t0 - c0 + c1 - 1); c3 += 16)
            corr[(t0 + c1) * 1200 + (c0 + c3)] = shared_corr_1[t0][c3];
      }
}
