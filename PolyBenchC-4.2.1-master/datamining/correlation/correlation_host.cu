#include <assert.h>
#include <stdio.h>
#include "correlation_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* correlation.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "correlation.h"


/* Array initialization. */
static
void init_array (int m,
		 int n,
		 DATA_TYPE *float_n,
		 DATA_TYPE POLYBENCH_2D(data,N,M,n,m))
{
  int i, j;

  *float_n = (DATA_TYPE)N;

  for (i = 0; i < N; i++)
    for (j = 0; j < M; j++)
      data[i][j] = (DATA_TYPE)(i*j)/M + i;

}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m,
		 DATA_TYPE POLYBENCH_2D(corr,M,M,m,m))

{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("corr");
  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
      if ((i * m + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
      fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, corr[i][j]);
    }
  POLYBENCH_DUMP_END("corr");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_correlation(int m, int n,
			DATA_TYPE float_n,
			DATA_TYPE POLYBENCH_2D(data,N,M,n,m),
			DATA_TYPE POLYBENCH_2D(corr,M,M,m,m),
			DATA_TYPE POLYBENCH_1D(mean,M,m),
			DATA_TYPE POLYBENCH_1D(stddev,M,m))
{
  int i, j, k;

  DATA_TYPE eps = SCALAR_VAL(0.1);


  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_corr;
    double *dev_data;
    double *dev_mean;
    double *dev_stddev;
    
    cudaCheckReturn(hipMalloc((void **) &dev_corr, (m) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_data, (n) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_mean, (m) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_stddev, (m) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev_corr, corr, (m) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    if (n >= 1)
      cudaCheckReturn(hipMemcpy(dev_data, data, (n) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    if (m >= 2)
      {
        dim3 k0_dimBlock(32);
        dim3 k0_dimGrid(ppcg_min(32768, (m + 30) / 32));
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_corr, m, n);
        cudaCheckKernel();
      }
      
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(ppcg_min(32768, (m + 31) / 32));
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_data, dev_mean, m, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k2_dimBlock(32);
      dim3 k2_dimGrid(ppcg_min(32768, (m + 31) / 32));
      kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_data, float_n, dev_mean, dev_stddev, m, n);
      cudaCheckKernel();
    }
    
    {
      dim3 k3_dimBlock(32);
      dim3 k3_dimGrid(ppcg_min(32768, (m + 31) / 32));
      kernel3 <<<k3_dimGrid, k3_dimBlock>>> (eps, float_n, dev_stddev, m, n);
      cudaCheckKernel();
    }
    
    if (n >= 1)
      {
        dim3 k4_dimBlock(16, 32);
        dim3 k4_dimGrid(ppcg_min(256, (m + 31) / 32), ppcg_min(256, (n + 31) / 32));
        kernel4 <<<k4_dimGrid, k4_dimBlock>>> (dev_data, float_n, dev_mean, dev_stddev, m, n);
        cudaCheckKernel();
      }
      
    if (m >= 2) {
      {
        dim3 k5_dimBlock(16, 32);
        dim3 k5_dimGrid(ppcg_min(256, (m + 30) / 32), ppcg_min(256, (m + 30) / 32));
        kernel5 <<<k5_dimGrid, k5_dimBlock>>> (dev_corr, m, n);
        cudaCheckKernel();
      }
      
      {
        dim3 k6_dimBlock(16, 32);
        dim3 k6_dimGrid(ppcg_min(256, (m + 31) / 32), ppcg_min(256, (m + 30) / 32));
        kernel6 <<<k6_dimGrid, k6_dimBlock>>> (dev_corr, dev_data, m, n);
        cudaCheckKernel();
      }
      
    }
    cudaCheckReturn(hipMemcpy(corr, dev_corr, (m) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    if (n >= 1)
      cudaCheckReturn(hipMemcpy(data, dev_data, (n) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(mean, dev_mean, (m) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(stddev, dev_stddev, (m) * sizeof(double), hipMemcpyDeviceToHost));
    corr[m - 1][m - 1] = 1.;
    cudaCheckReturn(hipFree(dev_corr));
    cudaCheckReturn(hipFree(dev_data));
    cudaCheckReturn(hipFree(dev_mean));
    cudaCheckReturn(hipFree(dev_stddev));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  DATA_TYPE float_n;
  POLYBENCH_2D_ARRAY_DECL(data,DATA_TYPE,N,M,n,m);
  POLYBENCH_2D_ARRAY_DECL(corr,DATA_TYPE,M,M,m,m);
  POLYBENCH_1D_ARRAY_DECL(mean,DATA_TYPE,M,m);
  POLYBENCH_1D_ARRAY_DECL(stddev,DATA_TYPE,M,m);

  /* Initialize array(s). */
  init_array (m, n, &float_n, POLYBENCH_ARRAY(data));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_correlation (m, n, float_n,
		      POLYBENCH_ARRAY(data),
		      POLYBENCH_ARRAY(corr),
		      POLYBENCH_ARRAY(mean),
		      POLYBENCH_ARRAY(stddev));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, POLYBENCH_ARRAY(corr)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(data);
  POLYBENCH_FREE_ARRAY(corr);
  POLYBENCH_FREE_ARRAY(mean);
  POLYBENCH_FREE_ARRAY(stddev);

  return 0;
}
